
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<sys/time.h>
#define N 99999999
__global__ void piv2(double* aread)
{
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    double dx=1.0/N , y, x=0.0;
    if(i<N)
    {
       x=i*dx;
		y = sqrt(1-x*x);  
		
        aread[i]=dx*y;
    }    	
}
int main()
{
	int i;
	double area=0.0, pi,*aread,*arr; 
	double exe_time;
	struct timeval stop_time, start_time;
    gettimeofday(&start_time, NULL);

	arr=(double *)malloc(N*sizeof(double));
    hipMalloc(&aread,N*sizeof(double));
    
    int numperBlocks=256;     
    int num_block= (N/256)+1; //
   piv2<<<num_block,numperBlocks >>>(aread);
   hipMemcpy(arr,aread,N*sizeof(double),hipMemcpyDeviceToHost);
   for(i=0;i<N;i++)
   {
    area+=arr[i];
   }
	gettimeofday(&stop_time, NULL);	
	exe_time = (stop_time.tv_sec+(stop_time.tv_usec/1000000.0)) - (start_time.tv_sec+(start_time.tv_usec/1000000.0));
	pi = 4.0*area;
	printf("\n Value of pi is = %.16lf\n Execution time is = %lf seconds\n", pi, exe_time);
    hipFree(aread);
    free(arr);
	return 0;
}

