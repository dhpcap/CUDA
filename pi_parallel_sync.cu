
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<sys/time.h>
#define N 99999999
#define NUM_THDS 256
__global__ void piv2(double* aread)
{
   
    int i=blockIdx.x*blockDim.x+threadIdx.x;
     int j;
    double dx=1.0/N , y, x=0.0;
    double tmp =0.0;
 __shared__ double tmp_area[NUM_THDS];
 tmp_area[threadIdx.x]=0.0;
    if(i<N)
    {
       x=i*dx;
		y = sqrt(1-x*x);  
		
        tmp_area[threadIdx.x]=dx*y;
    } 
__syncthreads();
if(i<N)
{
    if(threadIdx.x==0)
    {
        tmp =0.0;
       for(j=0;j<NUM_THDS;j++)
        {
          tmp+=tmp_area[j];//tmp_area[threadIdx.x];
       }
      aread[blockIdx.x]=tmp;
    }
}   	
}

int main()
{
	int i;
	double area=0.0, pi,*aread,*arr; 
	double exe_time;
	struct timeval stop_time, start_time;
    gettimeofday(&start_time, NULL);
    int numperBlocks=NUM_THDS;     
    int num_block= (N/256)+1;

	arr=(double *)malloc (num_block*sizeof(double));
    hipMalloc(&aread,num_block*sizeof(double));
    
     //
   piv2<<<num_block,numperBlocks >>>(aread);

   hipMemcpy(arr,aread,num_block*sizeof(double),hipMemcpyDeviceToHost);
   for(i=0;i<num_block;i++)
   {
    area+=arr[i];
   }
	gettimeofday(&stop_time, NULL);	
	exe_time = (stop_time.tv_sec+(stop_time.tv_usec/1000000.0)) - (start_time.tv_sec+(start_time.tv_usec/1000000.0));
	pi = 4.0*area;

	printf("\n Value of pi is = %.16lf\n Execution time is = %lf seconds\n", pi, exe_time);
    hipFree(aread);
    free(arr);
	return 0;
}

