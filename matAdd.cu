
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#define N 256
__global__ void MatAdd(float* A,float* B,float* C)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  int j=blockIdx.y*blockDim.y+threadIdx.y;
  C[i*N+j]=A[i*N+j]+B[i*N+j];
}

int main()
{
    int size=N*sizeof(float);
    int i,j;
    float a[N][N],b[N][N],c[N][N],*A,*B,*C;
    
    for(i=0;i<N;i++)
    {
        for(j=0;j<N;j++)
        {
            a[i][j]=5;
            b[i][j]=5;
            c[i][j]=0;
        }
    }

   hipMalloc(&A,size);
   hipMalloc(&B,size);
   hipMalloc(&C,size);
   hipMemcpy(A,a,size,hipMemcpyHostToDevice);
   hipMemcpy(B,b,size,hipMemcpyHostToDevice);
   //int numBlocks=1;
   dim3 numblocks(16,16);
   dim3 threadPerBlock(16,16);
   MatAdd<<<numblocks,threadPerBlock >>>(A,B,C);
   hipMemcpy(c,C,size,hipMemcpyDeviceToHost);

   for(i=0;i<N;i++)
    {
        for(j=0;j<N;j++)
        {
            printf("\t%f",c[i][j]);
        }
    }

    hipFree(A);
    hipFree(B);
    hipFree(C);

    return 0;
}