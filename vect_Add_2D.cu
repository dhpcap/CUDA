
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#define N 20
__global__ void arradd(int* md, int* nd, int* pd)
{
	int myid = threadIdx.y * blockDim.x + threadIdx.x;
	pd[myid] = md[myid] + nd[myid];
}
int main()
{
	int size = N *N* sizeof(int);
	int m[N][N], n[N][N], p[N][N],*md, *nd,*pd;
	int i=0,j=0;	
	for(i=0; i<N; i++ )
	{
		for(j=0; j<N; j++ )
		{
			m[i][j] = i;
			n[i][j] = i;
			p[i][j] = 0;
		}
	}
	hipMalloc(&md, size);
	hipMemcpy(md, m, size, hipMemcpyHostToDevice);
	hipMalloc(&nd, size);
	hipMemcpy(nd, n, size, hipMemcpyHostToDevice);
	hipMalloc(&pd, size);
	dim3   DimGrid(1, 1);     
	dim3   DimBlock(N, N);   
	arradd<<< DimGrid,DimBlock >>>(md,nd,pd);
	hipMemcpy(p, pd, size, hipMemcpyDeviceToHost);
	hipFree(md); 
	hipFree(nd);
	hipFree (pd);
	for(i=0; i<N; i++ )
	{
		for(j=0; j<N; j++ )
		{
			printf("\t%d",p[i][j]);
		}
		printf("\n");
	}
	hipFree(md);
	hipFree(nd);
	hipFree(pd);
	return 0;
}
