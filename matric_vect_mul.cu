
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>
#define VECTORSIZE 10
__global__ void matVecMul(int* A,int* B, int* C)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  int j;
  int sum=0;
  if(i<VECTORSIZE)
  {
    sum = 0;
		for(j=0;j<VECTORSIZE;j++)
		{
			sum = sum + A[i*VECTORSIZE+j]*B[i];	
		}
		C[i] =  sum;
  }
}
int main(int argc, char **argv)
{
	int size=VECTORSIZE * sizeof(int);
	int i, j;
	int *A, *B, *C,*Ad,*Bd,*Cd;		
	double exe_time;
	struct timeval stop_time, start_time;
	
	//Allocate the arrays
	A = (int *)malloc(VECTORSIZE*VECTORSIZE*sizeof(int));
	B = (int *)malloc(VECTORSIZE*sizeof(int));
	C = (int *)malloc(VECTORSIZE*sizeof(int));
	
	//Initialize data to some value
	for(i=0;i<VECTORSIZE;i++)
	{
		for(j=0;j<VECTORSIZE;j++)
		{
			A[i*VECTORSIZE+j] = 1;	
		}
		B[i] = 1;
	}
	
    // for serial code
	//print the data
	/*printf("\nInitial data: \n");
	for(i=0;i<VECTORSIZE;i++)
	{
		for(j=0;j<VECTORSIZE;j++)                
		{
			printf("\t%d ", A[i*VECTORSIZE+j]);	
		}
		printf("\n");
	}
	for(i=0;i<VECTORSIZE;i++)
	{
		printf("\t%d", B[i]);
	}	*/
	
	gettimeofday(&start_time, NULL);
     
     hipMalloc(&Ad, VECTORSIZE*size);
     hipMalloc(&Bd, size);
     hipMalloc(&Cd, size);

    hipMemcpy(Ad, A ,VECTORSIZE*VECTORSIZE*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(Bd, B ,VECTORSIZE*sizeof(int),hipMemcpyHostToDevice);
    dim3   DimGrid(1, 1);     
	dim3   DimBlock(VECTORSIZE,1);
    matVecMul<<<DimGrid,DimBlock >>>(Ad,Bd,Cd);
    hipMemcpy(C,Cd,VECTORSIZE*sizeof(int),hipMemcpyDeviceToHost);
	
    // for serial code
	/*for(i=0;i<VECTORSIZE;i++)
	{
		sum = 0;
		for(j=0;j<VECTORSIZE;j++)
		{
			sum = sum + A[i*VECTORSIZE+j]*B[i];	
		}
		C[i] =  sum;
	}*/
	
	gettimeofday(&stop_time, NULL);	
	exe_time = (stop_time.tv_sec+(stop_time.tv_usec/1000000.0)) - (start_time.tv_sec+(start_time.tv_usec/1000000.0));
	
	//print the data
	printf("\nMatrix & Vector multiplication output: \n");
	for(i=0;i<VECTORSIZE;i++)
	{
		printf("\t%d", C[i]);	
	}
	printf("\n Execution time is = %lf seconds\n", exe_time);
	
	printf("\nProgram exit!\n");
	
	//Free arrays
	free(A); 
	free(B);
	free(C);
    return 0;
}
