
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#define N 200
__global__ void vacAdd(float* A, float* B, float* C)
{
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    C[i]=A[i]+B[i];

}

int main()
{
    int size=N*sizeof(float);
    float a[N],b[N],c[N],*A,*B,*C;
    int i;
    
    for(i=0;i<N;i++)
    {
        a[i]=i;
        b[i]=i;
        c[i]=0;
    }
    hipMalloc(&A,size);
    hipMalloc(&B,size);
    hipMalloc(&C,size);
    hipMemcpy(A,a,size,hipMemcpyHostToDevice);
    hipMemcpy(B,b,size,hipMemcpyHostToDevice);

    vacAdd<<<1,N>>>(A,B,C);

    hipMemcpy(c,C,size,hipMemcpyDeviceToHost);
    for(i=0;i<N;i++)
    {
        printf("\t%f",c[i]);
    }
    hipFree(A);
    hipFree(B);
    return 0;
}