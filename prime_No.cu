

#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<sys/time.h>
#define N 1000000
/*
                N  PRIME_NUMBER

                1           0
               10           4
              100          25
            1,000         168
           10,000       1,229
          100,000       9,592
        1,000,000      78,498
       10,000,000     664,579
      100,000,000   5,761,455
    1,000,000,000  50,847,534

*/
__global__ void prime(int* countd)
{
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    int j;
    int flag = 0;
    if((i>2) && (i<N))
    {
		for(j=2;j<i;j++)	
	    	{
		    if((i%j) == 0)
		    {
			    flag = 1;
			    break;
		    }
	    	}
        	if(flag == 0)
        	{
            		countd[i]=1;
        	}
            else
            {
              countd[i]=0;  
            }
    }        
}

int main()
{
	int i;
	int count,*countd,*cnt;
	double exe_time;
	struct timeval stop_time, start_time;
    int size =  N*sizeof(int);
	
	count = 1; // 2 is prime. Our loop starts from 3
	
	gettimeofday(&start_time, NULL);

    cnt=(int *)malloc(N*sizeof(int));
    hipMalloc(&countd,N*sizeof(int));

    //cudaMalloc(&flagd,size);
    //cudaMemcpy(countd,cnt,size,cudaMemcpyHostToDevice);
    //    cudaMemcpy(flagd,flag,size,cudaMemcpyHostToDevice);

    int numBlocks=256;     
    int num_block= (N/numBlocks)+1; 
    prime<<<num_block,numBlocks >>>(countd);
	
    hipMemcpy(cnt, countd, size, hipMemcpyDeviceToHost);
	
	/*for(i=3;i<N;i++)
	{
	 	flag = 0;
		for(j=2;j<i;j++)	
	    	{
		    if((i%j) == 0)
		    {
			    flag = 1;
			    break;
		    }
	    	}
        	if(flag == 0)
        	{
            		count++;
        	}
	}*/
	
	gettimeofday(&stop_time, NULL);	
	exe_time = (stop_time.tv_sec+(stop_time.tv_usec/1000000.0)) - (start_time.tv_sec+(start_time.tv_usec/1000000.0));
	
    for(i=3;i<N;i++)
	{
	 	if(cnt[i]==1)
        {
            count++;
        }
    }    
	printf("\n Number of prime numbers = %d \n Execution time is = %lf seconds\n", count, exe_time);
	hipFree(countd); 
	hipFree(cnt);
    return 0;
}
