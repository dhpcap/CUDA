

#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<sys/time.h>
#define N 1000
#define NUM_THDS 256
/*
                N  PRIME_NUMBER

                1           0
               10           4
              100          25
            1,000         168
           10,000       1,229
          100,000       9,592
        1,000,000      78,498
       10,000,000     664,579
      100,000,000   5,761,455
    1,000,000,000  50,847,534

*/
__global__ void prime(int* countd)
{
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    int j;
    int flag = 0,count;
    __shared__ int tmp_count[NUM_THDS];
    tmp_count[threadIdx.x]=0;
    if((i>2) && (i<N))
    {
		for(j=2;j<i;j++)	
	    	{
		      if((i%j) == 0)
		       {
			    flag = 1;
			    break;
		       }
            }
	
        	if(flag == 0)
        	{
            	tmp_count[threadIdx.x]=1;
        	}
            else
            {
                tmp_count[threadIdx.x]=0;  
            }
       
   }     

__syncthreads();
if(i<N)
{
    if(threadIdx.x==0)
    {
        count=0;
       for(j=0;j<NUM_THDS;j++)
        {
          if(tmp_count[j]==1)
          count++;
       }
      countd[blockIdx.x]=count;
    }
}
}
int main()
{
	int i;
	int count,*countd,*cnt;
	double exe_time;
	struct timeval stop_time, start_time;
    int size =  NUM_THDS*sizeof(int);
	
	count = 1; // 2 is prime. Our loop starts from 3
	
	gettimeofday(&start_time, NULL);
    int numBlocks=NUM_THDS;     
	//int nunPerBlock=N; 
    int num_block= (N/numBlocks)+1; 


    cnt=(int *)malloc(num_block*sizeof(int));
    hipMalloc(&countd,num_block*sizeof(int));

    prime<<< num_block,numBlocks >>>(countd);
	
    hipMemcpy(cnt, countd, num_block*sizeof(int), hipMemcpyDeviceToHost);
	
	gettimeofday(&stop_time, NULL);	
	exe_time = (stop_time.tv_sec+(stop_time.tv_usec/1000000.0)) - (start_time.tv_sec+(start_time.tv_usec/1000000.0));
	
    for(i=0;i<num_block;i++)
	{
	 	
            count+=cnt[i];
       
    }    
	printf("\n Number of prime numbers = %d \n Execution time is = %lf seconds\n", count, exe_time);
	hipFree(countd); 
	free(cnt);
    return 0;
}
