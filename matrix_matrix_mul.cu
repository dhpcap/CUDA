
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>

#define VECTORSIZE 10000
#define NUM_THDS 256


__global__ void mat_mat_mul(int* Ad,int* Bd, int* Cd, int sum)
{
    int i,j,k;
    int myid=blockIdx.x*blockDim.x+threadIdx.x;
    i=myid/VECTORSIZE;
    j=myid%VECTORSIZE;
    if(myid<(VECTORSIZE*VECTORSIZE))
    {
		//for(j=0;j<VECTORSIZE;j++)
		//{
			sum = 0;
			for(k=0;k<VECTORSIZE;k++)
			{
				sum = sum + Ad[i*VECTORSIZE+k]*Bd[k*VECTORSIZE+j];	
			}
			Cd[i*VECTORSIZE+j] =  sum;
		//}
    }
}
int main(int argc, char **argv)
{
	int size;
	int i,j, sum;
	int *A, *B, *C,*Ad,*Bd,*Cd;		
	double exe_time;
	struct timeval stop_time, start_time;
	
	//Allocate and initialize the arrays
	A = (int *)malloc(VECTORSIZE*VECTORSIZE*sizeof(int));
	B = (int *)malloc(VECTORSIZE*VECTORSIZE*sizeof(int));
	C = (int *)malloc(VECTORSIZE*VECTORSIZE*sizeof(int));
	size= VECTORSIZE*(sizeof(int));
    for(i=0;i<VECTORSIZE;i++)
	{
		for(j=0;j<VECTORSIZE;j++)
		{
			A[i*VECTORSIZE+j] = 1;
			B[i*VECTORSIZE+j] = 2;
			C[i*VECTORSIZE+j] = 0;	
		}
	}
     gettimeofday(&start_time, NULL);
     hipMalloc(&Ad,VECTORSIZE*VECTORSIZE*sizeof(int));
     hipMalloc(&Bd,VECTORSIZE*VECTORSIZE*sizeof(int));
     hipMalloc(&Cd,VECTORSIZE*VECTORSIZE*sizeof(int));
     hipMemcpy(Ad,A,VECTORSIZE*VECTORSIZE*sizeof(int),hipMemcpyHostToDevice);
     hipMemcpy(Bd,B,VECTORSIZE*VECTORSIZE*sizeof(int),hipMemcpyHostToDevice);
     int numperBlocks=NUM_THDS;     
    int total_num_block=VECTORSIZE*VECTORSIZE;
    int num_blocks=total_num_block/numperBlocks+1;
     mat_mat_mul<<<num_blocks,numperBlocks >>>(Ad,Bd,Cd,sum);
	
    hipMemcpy(C, Cd, VECTORSIZE*VECTORSIZE*sizeof(int), hipMemcpyDeviceToHost);
	//Initialize data to some value
	
	
	//print the data
	/*printf("\nInitial data: \n");
	printf("\n A matrix:\n");
	for(i=0;i<VECTORSIZE;i++)
	{
		for(j=0;j<VECTORSIZE;j++)
		{
			printf("\t%d ", A[i*VECTORSIZE+j]);	
		}
		printf("\n");
	}
	printf("\n B matrix:\n");
	for(i=0;i<VECTORSIZE;i++)
	{
		for(j=0;j<VECTORSIZE;j++)
		{
			printf("\t%d ", B[i*VECTORSIZE+j]);	
		}
		printf("\n");
	}	
	*/
	
	
	/*for(i=0;i<VECTORSIZE;i++)
	{
		for(j=0;j<VECTORSIZE;j++)
		{
			sum = 0;
			for(k=0;k<VECTORSIZE;k++)
			{
				sum = sum + A[i*VECTORSIZE+k]*B[k*VECTORSIZE+j];	
			}
			C[i*VECTORSIZE+j] =  sum;
		}
	}*/
	
	gettimeofday(&stop_time, NULL);	
	exe_time = (stop_time.tv_sec+(stop_time.tv_usec/1000000.0)) - (start_time.tv_sec+(start_time.tv_usec/1000000.0));
	
	//print the data
	/*printf("\n C matrix:\n");
	for(i=0;i<VECTORSIZE;i++)
	{
		for(j=0;j<VECTORSIZE;j++)
		{
			printf("\t%d ", C[i*VECTORSIZE+j]);	
		}
		printf("\n");
	}*/
    printf("\nC[5*VECTORSIZE+5] = %d ", C[5*VECTORSIZE+5]);	
	printf("\n Execution time is = %lf seconds\n", exe_time);
	
	printf("\nProgram exit!\n");
	
	//Free arrays
	free(A); 
	free(B);
	free(C);
    hipFree(Ad);
    hipFree(Bd);
    hipFree(Cd);
}
